#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__  void AddInts(int* a, int* b)
{
//This could be any operation that takes more than 2 seconds
//Here I've rather pointlessly asked a single CUDA thread to
//add b to a 10,000,005 times.
for(int i=0;i<10000005;i++)
	a[0] += b[0];

}

int main()
{
int h_a = 0, h_b = 1; //Two integer variables
int *d_a, * d_b;	//GPU versions of the same

//Allocate space for copies of the integers on the GPU
if(hipMalloc((void**)&d_a, sizeof(int)) != hipSuccess) //We'll look errors later
{
	cout <<"Error allocating memory!"<<endl;
	return 0;

}
if(hipMalloc(&d_b, sizeof(int)) != hipSuccess)
{
	cout<<"Error allocating memory"<<endl;
	free(d_a);
	return 0;

}
//Copy the integer's values from the CPU to the GPU
if(hipMemcpy(d_a, &h_a, sizeof(int),hipMemcpyHostToDevice) != hipSuccess)
{
	cout<<"Error copying memory!"<<endl;
	hipFree(d_a);
	hipFree(d_b);
	return 0;
}

AddInts<<<1, 1>>>(d_a, d_b);

if(hipMemcpy(&h_a, d_a, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
{
cout<<"Error copying memory!"<<endl;
hipFree(d_a);
hipFree(d_b);
return 0;
}

cout<<"Adding 1 to 0 10,000,005 times gives"<<h_a<<endl;

hipFree(d_a);
hipFree(d_b);

hipDeviceReset();

return 0;
}