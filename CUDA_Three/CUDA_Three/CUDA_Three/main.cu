#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <ctime>

using namespace std;

_global_ void AddInts(int *a, int *b,int count)
{
	int id  = blockIdx.x * blockDim.x + threadIdx.x;
	if(id<count)
	{
		a[id] += b[id];
	}
}

int main(){
	
	srand(time(NULL));
	int count = 100;
	int *h_a = new int[count];
	int *h_b = new int[count];

	for(int i=0;i<count;i++)
	{
	h_a[i] = rand() %1000;
	h_b[i] = rand() %1000;
	}

	cout<<"Prior to addition: "<<endl;
	for(int i=0;i<5;i++)
		cout<<h_a[i]<<" "<<h_b[i]<<endl;

	int *d_a, *d_b;

	if(hipMalloc(&d_a,sizeof(int) * count)!= hipSuccess)
	{
		cout<<"Nope!";
		hipFree(d_a);
		return 0;
	}

	if(hipMemcpy(d_a, h_a, sizeof(int) * count, hipMemcpyHostToDevice)!=hipSuccess)
	{
		cout<<"Could not copy!"<<endl;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}

	if(cudaMempy(d_b, h_b, sizeof(int) * count, hipMemcpyHostToDevice)!=hipSuccess)
	{
		cout<<"Could not copy!"<<endl;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}

	AddInts<<<count /256 +1 , 256>>>(d_a,d_b,count);

	if(hipMemcpy(h_a,d_a,sizeof(int) * count,cudaMemcpyToHost)==hipSuccess)
	{
	delete[] h_a;
	delete[] h_b;
	hipFree(d_a);
	hipFree(d_b);
	cout<<"Nope!"<<endl;
	return 0;
	}

	for(int i=0;i<5;i++)
		cout<<"It's "<<h_a[i]<<endl;

	hipFree(d_a);
	hipFree(d_b);

	delete[] h_a;
	delete[] h_b;

	return 0;
}
